
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <iostream>
#include <fstream>
#include <sstream> 
#include <chrono>
#include <thread> 

#define LOGG true
#define BUFFER_SIZE 400

typedef unsigned char uint8_t;

typedef char* string_t;

#define total 1600
#define rate 1088

#define rounds 24

#define totalBytes total/8
#define rateBytes rate/8
#define rateBytesBuffer rateBytes*BUFFER_SIZE
#define rateLanes rate/64

#define delim_begin 0x06
#define delim_end 0x80

#define ROT(a, offset) ((((uint64_t)a) << offset) ^ (((uint64_t)a) >> (64-offset))) //credit
#define state (*State)
#define lane ((uint64_t*)state._8)

__device__ __constant__ uint8_t rho[25] =
    {0, 1, 62, 28, 27,
     36, 44, 6, 55, 20,
     3, 10, 43, 25, 39,
     41, 45, 15, 21, 8,
     18, 2, 61, 56, 14};

__device__ __constant__ uint8_t pi[25] =
    {0, 6, 12, 18, 24,
     3, 9, 10, 16, 22,
     1, 7, 13, 19, 20,
     4, 5, 11, 17, 23,
     2, 8, 14, 15, 21};

__device__ __constant__ uint64_t iota[24] =
  {
    0x0000000000000001UL, 0x0000000000008082UL,0x800000000000808aUL, 0x8000000080008000UL,
    0x000000000000808bUL, 0x0000000080000001UL,0x8000000080008081UL, 0x8000000000008009UL,
    0x000000000000008aUL, 0x0000000000000088UL,0x0000000080008009UL, 0x000000008000000aUL,
    0x000000008000808bUL, 0x800000000000008bUL,0x8000000000008089UL, 0x8000000000008003UL,
    0x8000000000008002UL, 0x8000000000000080UL,0x000000000000800aUL, 0x800000008000000aUL,
    0x8000000080008081UL, 0x8000000000008080UL,0x0000000080000001UL, 0x8000000080008008UL
};

union INTER{
    uint64_t _64[25];
    uint8_t _8[200];
};

__global__ void Keccak(string_t input, int size, union INTER* State){
    union INTER iState; 
    int buffer_i = 0;
    while(size>0 && (buffer_i++) < BUFFER_SIZE){
        if(size<rateBytes){    
            for(int i=0; i<size; i++)
                state._8[i] ^= input[i];
            state._8[size] ^= delim_begin;//padding
            state._8[rateBytes - 1] ^= delim_end;
        }
        else{
            for(int i=0; i<rateLanes; i++){
                state._64[i] ^= ((uint64_t*)input)[i];
            }   
            input += rateBytes;
        }
        
        __shared__ uint64_t CrossPlane[5];
        uint64_t D;

        uint8_t i = threadIdx.x;
        uint8_t i_5 = i*blockDim.x;

        for(int round = 0; round<rounds; round++){
            //Omega
        CrossPlane[i] = state._64[i] ^ state._64[i + 5] ^ state._64[i + 10] ^ state._64[i + 15] ^ state._64[i + 20];
        //__syncthreads();

        D = CrossPlane[i==0?4:(i-1)] ^ ROT(CrossPlane[i==4?0:(i+1)], 1);
        for(int y = 0; y<5; y++)
            {state._64[i + y*5] ^= D;}
        //__syncthreads();

        int indx = 0;
        //Rho and Pi
        for(int x = 0; x<5; x++){
            indx = pi[i_5+x];
            iState._64[i_5+x] = ROT(state._64[indx], rho[indx]);
        }
        //__syncthreads();

        //Chi
        for(int x = 0; x<5; x++)
        {
            state._64[x+i_5] = iState._64[x+i_5] ^ (~iState._64[(x+1)%5 +i_5] & iState._64[(x+2)%5 +i_5]);
        }
        //__syncthreads();

        state._64[0] ^= iota[round];
        __syncthreads();
        }
        size -= rateBytes;
    }
}

std::ofstream myfile;

void hostr(const char* path, int times=1){
    for(int tm = 0; tm<times; tm++){
        std::streamsize size;// = message2.length();
        std::streamsize sizeInit;
        char* contents = new char[rateBytesBuffer];
        std::ifstream istr(path, std::ios::in | std::ios::binary);
        std::streambuf* pbuf = NULL;

        if (istr) 
        {
            pbuf = istr.rdbuf();
            size = pbuf->pubseekoff(0, istr.end);
            sizeInit = size;
            std::cout << " File size is: " << size << "\n";
            pbuf->pubseekoff(0, istr.beg);       // rewind 
        }
        unsigned char output[32] = { "" };

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        char* input_d1;
        hipMalloc((void**)&input_d1, rateBytesBuffer);

        char* input_d2;
        hipMalloc((void**)&input_d2, rateBytesBuffer);

        union INTER* State_d;
        hipMalloc((void**)&State_d, sizeof(union INTER));
        hipMemset(State_d, 0, 200);

        hipStream_t stream_c;
        hipStream_t stream_k;
        hipStreamCreate(&stream_c);
        hipStreamCreate(&stream_k);
        
        int itter = 0;
        double hashed = pbuf->sgetn(contents, rateBytesBuffer);
        hipMemcpyAsync((itter?input_d2:input_d1), contents, size>rateBytesBuffer?rateBytesBuffer:size, hipMemcpyHostToDevice, stream_c);
        size -= rateBytesBuffer;

        for(int offset = 0; size>0; itter = !itter){
            Keccak<<<1,5,0,stream_k>>>((itter?input_d2:input_d1), size+rateBytesBuffer, State_d);
            double hashed = pbuf->sgetn(contents, rateBytesBuffer);
            size -= rateBytesBuffer;
            hipMemcpyAsync((itter?input_d1:input_d2), contents, rateBytesBuffer, hipMemcpyHostToDevice, stream_c);  
            hipStreamSynchronize(stream_k);
        }
        Keccak<<<1,5,0,stream_c>>>((itter?input_d2:input_d1), size+rateBytesBuffer, State_d); 
        hipMemcpyAsync(output, State_d, 32 * sizeof(char), hipMemcpyDeviceToHost, stream_c); 
        hipStreamSynchronize(stream_c);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        printf(" %s - ", path);
        std::cout << "elapsed time: " << milliseconds/1000 << "s\n";
        if(LOGG)myfile << path << "; " << milliseconds/1000 << "\n" ;

        for (int i = 0; i < 32; i++)
            printf("%02x", output[i]);

        hipFree(input_d1);
        hipFree(input_d2);
        hipFree(State_d);
        printf("\n\n");
        istr.close();
        //if(LOGG)myfile << path << "; " << elapsed_seconds.count() << "\n" ;
    }
}

int main( int argc, char *argv[] ){
    if(LOGG)myfile.open ("b_2s5t.csv");
    for(int pt = 1; pt<argc; pt++)
        hostr(argv[pt],1);
    if(LOGG)myfile << "\n" ;
    if(LOGG)myfile.close();
}